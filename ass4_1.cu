
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
//nvcc ass4_1.cu -o ass4_1  -lcuda
//./ass4_1
// Kernel function to add two vectors
__global__ void vectorAdd(int *a, int *b, int *c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    // Size of vectors
    int size = 10;

    // Host vectors
    int *h_a, *h_b, *h_c;
    h_a = new int[size];
    h_b = new int[size];
    h_c = new int[size];

    // Initialize input vectors
    for (int i = 0; i < size; ++i) {
        // int a;
        // std::cin>>a;
        // int b;
        // std::cin>>b;
        h_a[i] = i;
        h_b[i] = i*2;
    }
    std::cout << "1st vector" << std::endl;
    for (int i = 0; i < size; ++i) {
        std::cout << h_a[i] << " ";
    }
    std::cout << std::endl;
    std::cout << "2nd vector" << std::endl;
    for (int i = 0; i < size; ++i) {
        std::cout << h_b[i] << " ";
    }
    std::cout << std::endl;

    // Device vectors
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size * sizeof(int));
    hipMalloc((void **)&d_b, size * sizeof(int));
    hipMalloc((void **)&d_c, size * sizeof(int));

    // Copy input vectors from host to device memory
    hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block size
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    // Print kernel launch configuration
    std::cout << "Grid size: " << numBlocks << ", Block size: " << blockSize << std::endl;
    hipEvent_t start,stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    // Launch kernel
    vectorAdd<<<numBlocks, blockSize>>>(d_a, d_b, d_c, size);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // Copy result from device to host memory
    hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);
    // Verify result
    for (int i = 0; i < size; ++i) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            std::cerr << "Error: Incorrect result at index " << i << std::endl;
            break;
        }
    }
        // Print the resultant array
        std::cout << "Resultant array after vector addition:" << std::endl;
        for (int i = 0; i < size; ++i) {
            std::cout << h_c[i] << " ";
        }
        std::cout << std::endl;
        cout << "GPU result:\n";
        cout<<"Elapsed Time = "<<elapsedTime<<" milliseconds" << endl;
    // Clean up
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    std::cout << "Vector addition completed successfully!" << std::endl;

    return 0;
}
